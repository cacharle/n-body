#include "hip/hip_runtime.h"
#include "body.h"
#include <string.h>
#include <stdio.h>

#define CUDA_CHECK(x) do {                                                      \
        hipError_t cuda_check_result;                                          \
        if ((cuda_check_result = (x)) != hipSuccess)                           \
        {                                                                       \
            fprintf(stderr, "Error CUDA: %s:%d: %s\n\t%s",                      \
                __FILE__, __LINE__, hipGetErrorString(cuda_check_result), #x); \
            exit(EXIT_FAILURE);                                                 \
        }                                                                       \
    } while(0);

const float time_step = 0.001f;

// __global__ void bodies_gravitational_force_update(
//     struct body *bodies,
//     struct body *bodies_origin,
//     size_t bodies_count,
//     float gravity
// ) {
//     size_t b1_id = blockIdx.x * blockDim.x + threadIdx.x;
//     size_t b2_id = blockIdx.y * blockDim.y + threadIdx.y;
//     if (b1_id >= bodies_count || b2_id >= bodies_count)
//         return;
//     struct body *b1 = &bodies_origin[b1_id];
//     struct body *b2 = &bodies_origin[b2_id];
//     if (fabsf(b1->x - b2->x) < 0.01f || fabsf(b1->y - b2->y) < 0.01f)
//         return;
//     float distance_x = b1->x - b2->x;
//     float distance_y = b1->y - b2->y;
//     float distance_square = distance_x * distance_x + distance_y * distance_y;
//     float force = (b1->mass * b2->mass * gravity) / distance_square;
//     float dx = b1->x - b2->x;
//     float dy = b1->y - b2->y;
//     float magnitude_inverse = 1.0 / sqrt(dx * dx + dy * dy);
//     dx *= magnitude_inverse;
//     dy *= magnitude_inverse;
//     dx *= force;
//     dy *= force;
//
//     if (!isnan(dx) && !isnan(dy))
//     {
//         bodies[b1_id].acceleration_x += dx;
//         bodies[b1_id].acceleration_y += dy;
//     }
// }
//
// __global__ void update_bodies_position(struct body *bodies)
// {
//     size_t i = blockIdx.x * blockDim.x + threadIdx.x;
//     bodies[i].acceleration_x /= bodies[i].mass;
//     bodies[i].acceleration_y /= bodies[i].mass;
//     bodies[i].velocity_y -= bodies[i].acceleration_y * time_step;
//     bodies[i].velocity_x -= bodies[i].acceleration_x * time_step;
//     bodies[i].x += bodies[i].velocity_x * time_step;
//     bodies[i].y += bodies[i].velocity_y * time_step;
// }

// extern "C" void update_bodies_naive(struct body *bodies_host, size_t bodies_count, float gravity) {
//     for (size_t i = 0; i < bodies_count; i++)
//     {
//         bodies_host[i].acceleration_x = 0.0;
//         bodies_host[i].acceleration_y = 0.0;
//     }
//
//     size_t bodies_bytes = sizeof(struct body) * bodies_count;
//     if (bodies == NULL || bodies_origin == NULL)
//     {
//         CUDA_CHECK(hipMalloc(&bodies, bodies_bytes));
//         CUDA_CHECK(hipMalloc(&bodies_origin, bodies_bytes));
//     }
//     CUDA_CHECK(hipMemcpy(bodies, bodies_host, bodies_bytes, hipMemcpyHostToDevice));
//     CUDA_CHECK(hipMemcpy(bodies_origin, bodies, bodies_bytes, hipMemcpyDeviceToDevice));
//
//     size_t threads_count = 32;
//     size_t blocks_count = (bodies_count + threads_count - 1) / threads_count;
//     dim3 threads_dim(threads_count, threads_count);
//     dim3 blocks_dim(blocks_count, blocks_count);
//     bodies_gravitational_force_update<<<blocks_dim, threads_dim>>>(
//         bodies,
//         bodies_origin,
//         bodies_count,
//         gravity
//     );
//     hipDeviceSynchronize();
//     threads_count = 256;
//     blocks_count = (bodies_count + threads_count - 1) / threads_count;
//     threads_dim = dim3(threads_count);
//     blocks_dim = dim3(blocks_count);
//     update_bodies_position<<<blocks_dim, threads_dim>>>(bodies);
//
//     CUDA_CHECK(hipMemcpy(bodies_host, bodies, bodies_bytes, hipMemcpyDeviceToHost));
// }

// #include <thrust/extrema.h>

__global__ void bodies_struct_to_arrays(
    struct body *bodies,
    float *bodies_x,
    float *bodies_y,
    float *bodies_velocity_x,
    float *bodies_velocity_y)
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    bodies_x[idx] = bodies[idx].x;
    bodies_y[idx] = bodies[idx].y;
    bodies_velocity_x[idx] = bodies[idx].velocity_x;
    bodies_velocity_y[idx] = bodies[idx].velocity_y;
}

#define THREADS_COUNT 256

__global__ void min_max_element_kernel(float *xs, size_t count, float *mins, float *maxs)
{
    __shared__ float partial_min[THREADS_COUNT];
    __shared__ float partial_max[THREADS_COUNT];
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    // Current minimum of the binary tree level (start of the level range)
    partial_min[threadIdx.x] = (idx < count) ? xs[idx] : INFINITY;
    partial_max[threadIdx.x] = (idx < count) ? xs[idx] : -INFINITY;
    __syncthreads();
    // Compare the current element with the powers of 2 (min/max solved by another thread)
    // for (int stride = 1; stride < blockDim.x; stride <<= 1)
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) // Better for data locality than the above?
    {
        // if (threadIdx.x % stride == 0) {
        if (threadIdx.x < stride) {
            partial_min[threadIdx.x] = min(partial_min[threadIdx.x], partial_min[threadIdx.x + stride]);
            partial_max[threadIdx.x] = max(partial_max[threadIdx.x], partial_max[threadIdx.x + stride]);
        }
        __syncthreads();
    }
    // Set the min/max element in the *global* memory, the first thread of the block ends up with
    // the min/max element
    if (threadIdx.x == 0)
    {
        mins[blockIdx.x] = partial_min[0];
        maxs[blockIdx.x] = partial_max[0];
    }
    __syncthreads();
    // Lastly the first thread of the first block computes the min/max of all the other blocks
    if (blockIdx.x == 0 && threadIdx.x == 0)
    {
        for (size_t i = 1; i < gridDim.x; i++)
        {
            mins[0] = min(mins[0], mins[i]);
            maxs[0] = max(maxs[0], maxs[i]);
        }
    }
}

void min_max_element(float *xs, size_t count, float *pmin, float *pmax)
{
    size_t blocks_count = (count + THREADS_COUNT - 1) / THREADS_COUNT;
    dim3 threads_dim = dim3(THREADS_COUNT);
    dim3 blocks_dim = dim3(blocks_count);
    float *mins, *maxs;
    hipMalloc(&mins, sizeof(float) * blocks_count);
    hipMalloc(&maxs, sizeof(float) * blocks_count);
    min_max_element_kernel<<<blocks_dim, threads_dim>>>(xs, count, mins, maxs);
    hipMemcpy(pmin, mins, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(pmax, maxs, sizeof(float), hipMemcpyDeviceToHost);
    hipFree(mins);
    hipFree(maxs);
}

extern "C" void update_bodies_barnes_hut(struct body *bodies_host, size_t bodies_count, float gravity)
{

    static struct body *bodies = NULL, *bodies_origin = NULL;
    static float *bodies_x = NULL, *bodies_y = NULL, *bodies_velocity_x = NULL, *bodies_velocity_y = NULL;

    size_t bodies_bytes = sizeof(struct body) * bodies_count;
    if (bodies == NULL || bodies_origin == NULL)
    {
        CUDA_CHECK(hipMalloc(&bodies, bodies_bytes));
        CUDA_CHECK(hipMalloc(&bodies_origin, bodies_bytes));

        CUDA_CHECK(hipMalloc(&bodies_x, bodies_count * sizeof(float)));
        CUDA_CHECK(hipMalloc(&bodies_y, bodies_count * sizeof(float)));
        CUDA_CHECK(hipMalloc(&bodies_velocity_x, bodies_count * sizeof(float)));
        CUDA_CHECK(hipMalloc(&bodies_velocity_y, bodies_count * sizeof(float)));
    }

    CUDA_CHECK(hipMemcpy(bodies, bodies_host, bodies_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(bodies_origin, bodies, bodies_bytes, hipMemcpyDeviceToDevice));

    hipDeviceSynchronize();

    size_t blocks_count = (bodies_count + THREADS_COUNT - 1) / THREADS_COUNT;
    // printf("%zu*%zu = %zu\n", threads_count, blocks_count, threads_count * blocks_count);
    dim3 threads_dim = dim3(THREADS_COUNT);
    dim3 blocks_dim = dim3(blocks_count);
    bodies_struct_to_arrays<<<blocks_dim, threads_dim>>>(
        bodies,
        bodies_x,
        bodies_y,
        bodies_velocity_x,
        bodies_velocity_y
    );

    float x_min, x_max, y_min, y_max;
    min_max_element(bodies_x, bodies_count, &x_min, &x_max);
    min_max_element(bodies_y, bodies_count, &y_min, &y_max);
    printf("%.2f, %.2f -> %.2f, %.2f\n", x_min, y_min, x_max, y_max);


}


