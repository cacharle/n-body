#include "hip/hip_runtime.h"
#include "body.h"

__global__ void bodies_gravitational_force_update(struct body *bodies, struct body *bodies_origin, float gravity) {
    struct body *b1 = &bodies_origin[blockIdx.x];
    struct body *b2 = &bodies_origin[threadIdx.x];
    if (fabsf(b1->x - b2->x) < 0.01f || fabsf(b1->y - b2->y) < 0.01f)
        return;
    float distance_x = b1->x - b2->x;
    float distance_y = b1->y - b2->y;
    float distance_square = distance_x * distance_x + distance_y * distance_y;
    float force = (b1->mass * b2->mass * gravity) /
                  distance_square;  // maybe we can remove the `b1->mass *` because we end up
                                    // dividing by it at the end
    float dx = b1->x - b2->x;
    float dy = b1->y - b2->y;
    float magnitude_inverse = 1.0 / sqrt(dx * dx + dy * dy);
    dx *= magnitude_inverse;
    dy *= magnitude_inverse;
    dx *= force;
    dy *= force;
}

inline
hipError_t checkCuda(hipError_t result)
{
    if (result != hipSuccess)
    {
        fprintf(stderr, "Error CUDA: %s\n", hipGetErrorString(result));
        exit(EXIT_FAILURE);
    }
    return result;
}

extern "C" void update_bodies_naive(struct body *bodies_cpu, size_t bodies_count, float gravity) {
    struct body *bodies, *bodies_origin;
    size_t bodies_bytes = sizeof(struct body) * bodies_count;
    checkCuda(hipMalloc(&bodies, bodies_bytes));
    checkCuda(hipMalloc(&bodies_origin, bodies_bytes));
    checkCuda(hipMemcpy(bodies, bodies_cpu, bodies_bytes, hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(bodies_origin, bodies, bodies_bytes, hipMemcpyDeviceToDevice));

    bodies_gravitational_force_update<<<1, 1>>>(bodies, bodies_origin, gravity);

    hipFree(bodies);
    hipFree(bodies_origin);
}
