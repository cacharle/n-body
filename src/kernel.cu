#include "hip/hip_runtime.h"
#include "body.h"

const float time_step = 0.001f;

__global__ void bodies_gravitational_force_update(struct body *bodies, struct body *bodies_origin, size_t bodies_count, float gravity) {
    size_t b1_id = blockIdx.x * blockDim.x + threadIdx.x;
    size_t b2_id = blockIdx.y * blockDim.y + threadIdx.y;
    if (b1_id >= bodies_count || b2_id >= bodies_count)
        return;
    struct body *b1 = &bodies_origin[b1_id];
    struct body *b2 = &bodies_origin[b2_id];
    if (fabsf(b1->x - b2->x) < 0.01f || fabsf(b1->y - b2->y) < 0.01f)
        return;
    float distance_x = b1->x - b2->x;
    float distance_y = b1->y - b2->y;
    float distance_square = distance_x * distance_x + distance_y * distance_y;
    float force = (b1->mass * b2->mass * gravity) / distance_square;
    float dx = b1->x - b2->x;
    float dy = b1->y - b2->y;
    float magnitude_inverse = 1.0 / sqrt(dx * dx + dy * dy);
    dx *= magnitude_inverse;
    dy *= magnitude_inverse;
    dx *= force;
    dy *= force;

    if (!isnan(dx) && !isnan(dy))
    {
        bodies[b1_id].acceleration_x += dx;
        bodies[b1_id].acceleration_y += dy;
    }
}

__global__ void update_bodies_position(struct body *bodies)
{
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    bodies[i].acceleration_x /= bodies[i].mass;
    bodies[i].acceleration_y /= bodies[i].mass;
    bodies[i].velocity_y -= bodies[i].acceleration_y * time_step;
    bodies[i].velocity_x -= bodies[i].acceleration_x * time_step;
    bodies[i].x += bodies[i].velocity_x * time_step;
    bodies[i].y += bodies[i].velocity_y * time_step;
}

#define CUDA_CHECK(x) do {                                                      \
        hipError_t cuda_check_result;                                          \
        if ((cuda_check_result = (x)) != hipSuccess)                           \
        {                                                                       \
            fprintf(stderr, "Error CUDA: %s:%d: %s\n\t%s",                      \
                __FILE__, __LINE__, hipGetErrorString(cuda_check_result), #x); \
            exit(EXIT_FAILURE);                                                 \
        }                                                                       \
    } while(0);

struct body *bodies = NULL, *bodies_origin = NULL;

extern "C" void update_bodies_naive(struct body *bodies_host, size_t bodies_count, float gravity) {
    for (size_t i = 0; i < bodies_count; i++)
    {
        bodies_host[i].acceleration_x = 0.0;
        bodies_host[i].acceleration_y = 0.0;
    }

    size_t bodies_bytes = sizeof(struct body) * bodies_count;
    if (bodies == NULL || bodies_origin == NULL)
    {
        CUDA_CHECK(hipMalloc(&bodies, bodies_bytes));
        CUDA_CHECK(hipMalloc(&bodies_origin, bodies_bytes));
    }
    CUDA_CHECK(hipMemcpy(bodies, bodies_host, bodies_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(bodies_origin, bodies, bodies_bytes, hipMemcpyDeviceToDevice));

    size_t threads_count = 32;
    size_t blocks_count = (bodies_count + threads_count - 1) / threads_count;
    dim3 threads_dim(threads_count, threads_count);
    dim3 blocks_dim(blocks_count, blocks_count);
    bodies_gravitational_force_update<<<blocks_dim, threads_dim>>>(bodies, bodies_origin, bodies_count, gravity);
    hipDeviceSynchronize();
    threads_count = 256;
    blocks_count = (bodies_count + threads_count - 1) / threads_count;
    threads_dim = dim3(threads_count);
    blocks_dim = dim3(blocks_count);
    update_bodies_position<<<blocks_dim, threads_dim>>>(bodies);

    CUDA_CHECK(hipMemcpy(bodies_host, bodies, bodies_bytes, hipMemcpyDeviceToHost));
}
