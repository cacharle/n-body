#include "hip/hip_runtime.h"
#include <cmath>
#include "body.h"

float quadtree_max_x = -INFINITY;
float quadtree_min_x = +INFINITY;
float quadtree_max_y = -INFINITY;
float quadtree_min_y = +INFINITY;

// __global__ void cu_quadtree_bounding_box(const struct body *bodies)
// {
//     size_t i = blockIdx.x * blockDim.x + threadIdx.x;
//     if (bodies[i].x > max_x)
//         atomicCAS((int*)&max_x, *(int*)&bodies[i].x, *(int*)&bodies[i].x);
//     if (bodies[i].x < min_x)
//         atomicCAS((int*)&min_x, *(int*)&bodies[i].x, *(int*)&bodies[i].x);
//     if (bodies[i].y > max_y)
//         atomicCAS((int*)&max_y, *(int*)&bodies[i].y, *(int*)&bodies[i].y);
//     if (bodies[i].y < min_y)
//         atomicCAS((int*)&min_y, *(int*)&bodies[i].y, *(int*)&bodies[i].y);
// }

// void init_quadtree_root(struct quadtree *root, const struct body *bodies)
// {
//     cu_quadtree_bounding_box<<<1, 1024>>>(bodies);
//     hipDeviceSynchronize();
//     root->start_x = quadtree_min_x;
//     root->end_x = quadtree_max_x;
//     root->start_y = quadtree_min_y;
//     root->end_y = quadtree_max_y;
// }
